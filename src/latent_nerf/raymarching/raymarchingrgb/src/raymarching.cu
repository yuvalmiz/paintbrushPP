#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}

inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}


////////////////////////////////////////////////////
/////////////           utils          /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// nears/fars: [N]
// scalar_t should always be float in use.
template <typename scalar_t>
__global__ void kernel_near_far_from_aabb(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const scalar_t * __restrict__ aabb,
    const uint32_t N,
    const float min_near,
    scalar_t * nears, scalar_t * fars
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // get near far (assume cube scene)
    float near = (aabb[0] - ox) * rdx;
    float far = (aabb[3] - ox) * rdx;
    if (near > far) swapf(near, far);

    float near_y = (aabb[1] - oy) * rdy;
    float far_y = (aabb[4] - oy) * rdy;
    if (near_y > far_y) swapf(near_y, far_y);

    if (near > far_y || near_y > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_y > near) near = near_y;
    if (far_y < far) far = far_y;

    float near_z = (aabb[2] - oz) * rdz;
    float far_z = (aabb[5] - oz) * rdz;
    if (near_z > far_z) swapf(near_z, far_z);

    if (near > far_z || near_z > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_z > near) near = near_z;
    if (far_z < far) far = far_z;

    if (near < min_near) near = min_near;

    nears[n] = near;
    fars[n] = far;
}


void near_far_from_aabb(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor aabb, const uint32_t N, const float min_near, at::Tensor nears, at::Tensor fars) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "near_far_from_aabb", ([&] {
        kernel_near_far_from_aabb<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), aabb.data_ptr<scalar_t>(), N, min_near, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>());
    }));
}


// rays_o/d: [N, 3]
// radius: float
// coords: [N, 2]
template <typename scalar_t>
__global__ void kernel_sph_from_ray(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const float radius,
    const uint32_t N,
    scalar_t * coords
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    coords += n * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // solve t from || o + td || = radius
    const float A = dx * dx + dy * dy + dz * dz;
    const float B = ox * dx + oy * dy + oz * dz; // in fact B / 2
    const float C = ox * ox + oy * oy + oz * oz - radius * radius;

    const float t = (- B + sqrtf(B * B - A * C)) / A; // always use the larger solution (positive)

    // solve theta, phi (assume y is the up axis)
    const float x = ox + t * dx, y = oy + t * dy, z = oz + t * dz;
    const float theta = atan2(sqrtf(x * x + z * z), y); // [0, PI)
    const float phi = atan2(z, x); // [-PI, PI)

    // normalize to [-1, 1]
    coords[0] = 2 * theta * RPI() - 1;
    coords[1] = phi * RPI();
}


void sph_from_ray(const at::Tensor rays_o, const at::Tensor rays_d, const float radius, const uint32_t N, at::Tensor coords) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "sph_from_ray", ([&] {
        kernel_sph_from_ray<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), radius, N, coords.data_ptr<scalar_t>());
    }));
}


// coords: int32, [N, 3]
// indices: int32, [N]
__global__ void kernel_morton3D(
    const int * __restrict__ coords,
    const uint32_t N,
    int * indices
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;
    indices[n] = __morton3D(coords[0], coords[1], coords[2]);
}


void morton3D(const at::Tensor coords, const uint32_t N, at::Tensor indices) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(coords.data_ptr<int>(), N, indices.data_ptr<int>());
}


// indices: int32, [N]
// coords: int32, [N, 3]
__global__ void kernel_morton3D_invert(
    const int * __restrict__ indices,
    const uint32_t N,
    int * coords
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;

    const int ind = indices[n];

    coords[0] = __morton3D_invert(ind >> 0);
    coords[1] = __morton3D_invert(ind >> 1);
    coords[2] = __morton3D_invert(ind >> 2);
}


void morton3D_invert(const at::Tensor indices, const uint32_t N, at::Tensor coords) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D_invert<<<div_round_up(N, N_THREAD), N_THREAD>>>(indices.data_ptr<int>(), N, coords.data_ptr<int>());
}


// grid: float, [C, H, H, H]
// N: int, C * H * H * H / 8
// density_thresh: float
// bitfield: uint8, [N]
template <typename scalar_t>
__global__ void kernel_packbits(
    const scalar_t * __restrict__ grid,
    const uint32_t N,
    const float density_thresh,
    uint8_t * bitfield
) {
    // parallel per byte
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grid += n * 8;

    uint8_t bits = 0;

    #pragma unroll
    for (uint8_t i = 0; i < 8; i++) {
        bits |= (grid[i] > density_thresh) ? ((uint8_t)1 << i) : 0;
    }

    bitfield[n] = bits;
}


void packbits(const at::Tensor grid, const uint32_t N, const float density_thresh, at::Tensor bitfield) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grid.scalar_type(), "packbits", ([&] {
        kernel_packbits<<<div_round_up(N, N_THREAD), N_THREAD>>>(grid.data_ptr<scalar_t>(), N, density_thresh, bitfield.data_ptr<uint8_t>());
    }));
}

////////////////////////////////////////////////////
/////////////         training         /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// grid: [CHHH / 8]
// xyzs, dirs, deltas: [M, 3], [M, 3], [M, 2]
// dirs: [M, 3]
// rays: [N, 3], idx, offset, num_steps
template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,  
    const uint8_t * __restrict__ grid,
    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M,
    const scalar_t* __restrict__ nears, 
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * deltas,
    int * rays,
    int * counter,
    const scalar_t* __restrict__ noises
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    // ray marching
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;
    
    float t0 = near;
    
    // perturb
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max) * noise;

    // first pass: estimation of num_steps
    float t = t0;
    uint32_t num_steps = 0;

    //if (t < far) printf("valid ray %d t=%f near=%f far=%f \n", n, t, near, far);
    
    while (t < far && num_steps < max_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        //if (n == 0) printf("t=%f density=%f vs thresh=%f step=%d\n", t, density, density_thresh, num_steps);

        if (occ) {
            num_steps++;
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;

            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }

    //printf("[n=%d] num_steps=%d, near=%f, far=%f, dt=%f, max_steps=%f\n", n, num_steps, near, far, dt_min, (far - near) / dt_min);

    // second pass: really locate and write points & dirs
    uint32_t point_index = atomicAdd(counter, num_steps);
    uint32_t ray_index = atomicAdd(counter + 1, 1);
    
    //printf("[n=%d] num_steps=%d, point_index=%d, ray_index=%d\n", n, num_steps, point_index, ray_index);

    // write rays
    rays[ray_index * 3] = n;
    rays[ray_index * 3 + 1] = point_index;
    rays[ray_index * 3 + 2] = num_steps;

    if (num_steps == 0) return;
    if (point_index + num_steps > M) return;

    xyzs += point_index * 3;
    dirs += point_index * 3;
    deltas += point_index * 2;

    t = t0;
    uint32_t step = 0;

    float last_t = t;

    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        // query grid
        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max); 
            } while (t < tt);
        }
    }
}

void march_rays_train(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor grid, const float bound, const float dt_gamma, const uint32_t max_steps, const uint32_t N, const uint32_t C, const uint32_t H, const uint32_t M, const at::Tensor nears, const at::Tensor fars, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor rays, at::Tensor counter, at::Tensor noises) {

    static constexpr uint32_t N_THREAD = 128;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), grid.data_ptr<uint8_t>(), bound, dt_gamma, max_steps, N, C, H, M, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), counter.data_ptr<int>(), noises.data_ptr<scalar_t>());
    }));
}


// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;

    // accumulate
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, t = 0, d = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        t += deltas[1]; // real delta
        d += weight * t;

        ws += weight;

        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;
}


void composite_rays_train_forward(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor weights_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    image += index * 3;
    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;
    grad_sigmas += offset;
    grad_rgbs += offset * 3;

    // accumulate
    uint32_t step = 0;

    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        ws += weight;

        T *= 1.0f - alpha;

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;

        // write grad_sigmas
        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) +
            grad_image[1] * (T * rgbs[1] - (g_final - g)) +
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            grad_weights_sum[0] * (1 - ws_final)
        );

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;

        step++;
    }
}


void composite_rays_train_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor image, const uint32_t M, const uint32_t N, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void kernel_march_rays(
    const uint32_t n_alive,
    const uint32_t n_step,
    const int* __restrict__ rays_alive,
    const scalar_t* __restrict__ rays_t,
    const scalar_t* __restrict__ rays_o,
    const scalar_t* __restrict__ rays_d,
    const float bound,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t C, const uint32_t H,
    const uint8_t * __restrict__ grid,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t* xyzs, scalar_t* dirs, scalar_t* deltas,
    const scalar_t* __restrict__ noises
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    const float noise = noises[n];

    // locate
    rays_o += index * 3;
    rays_d += index * 3;
    xyzs += n * n_step * 3;
    dirs += n * n_step * 3;
    deltas += n * n_step * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    float t = rays_t[index]; // current ray's t
    const float near = nears[index], far = fars[index];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (C - 1)) / H;

    // march for n_step steps, record points
    uint32_t step = 0;

    // introduce some randomness
    t += clamp(t * dt_gamma, dt_min, dt_max) * noise;

    float last_t = t;

    while (t < far && step < n_step) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1, level), bound);
        const float mip_rbound = 1 / mip_bound;

        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            // calc dt
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            // step
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;

        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do {
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}


void march_rays(const uint32_t n_alive, const uint32_t n_step, const at::Tensor rays_alive, const at::Tensor rays_t, const at::Tensor rays_o, const at::Tensor rays_d, const float bound, const float dt_gamma, const uint32_t max_steps, const uint32_t C, const uint32_t H, const at::Tensor grid, const at::Tensor near, const at::Tensor far, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor noises) {
    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays", ([&] {
        kernel_march_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), bound, dt_gamma, max_steps, C, H, grid.data_ptr<uint8_t>(), near.data_ptr<scalar_t>(), far.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), noises.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive,
    const uint32_t n_step,
    const float T_thresh,
    int* rays_alive,
    scalar_t* rays_t,
    const scalar_t* __restrict__ sigmas,
    const scalar_t* __restrict__ rgbs,
    const scalar_t* __restrict__ deltas,
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id

    // locate
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;

    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;

    scalar_t t = rays_t[0]; // current ray's t

    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];

    // accumulate
    uint32_t step = 0;
    while (step < n_step) {

        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /*
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        -->
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t += deltas[1]; // real delta
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
}


void composite_rays(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor weights, at::Tensor depth, at::Tensor image) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}



// ------------------------------


template <typename scalar_t>
__global__ void kernel_composite_rays_train_localization_int_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,
    const scalar_t * __restrict__ loc_prob,
    const scalar_t * __restrict__ style_rgbs,
    const scalar_t * __restrict__ beck_rgbs,  
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, 
    scalar_t * depth,
    scalar_t * weights_sum,
    scalar_t * image_origin,
    scalar_t * image_prob,
    scalar_t * image_style_origin,
    scalar_t * image_back_origin,
    scalar_t * image_localization_mix,
    scalar_t * image_style_mix,
    scalar_t * image_back_mix
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[index] = 0;
        depth[index] = 0;
        image_origin[index * 3] = 0;
        image_origin[index * 3 + 1] = 0;
        image_origin[index * 3 + 2] = 0;
        
        image_prob[index] = 0;

        image_style_origin[index * 3] = 0;
        image_style_origin[index * 3 + 1] = 0;
        image_style_origin[index * 3 + 2] = 0;

        image_back_origin[index * 3] = 0;
        image_back_origin[index * 3 + 1] = 0;
        image_back_origin[index * 3 + 2] = 0;

        image_localization_mix[index * 3] = 0;
        image_localization_mix[index * 3 + 1] = 0;
        image_localization_mix[index * 3 + 2] = 0;

        image_style_mix[index * 3] = 0;
        image_style_mix[index * 3 + 1] = 0;
        image_style_mix[index * 3 + 2] = 0;

        image_back_mix[index * 3] = 0;
        image_back_mix[index * 3 + 1] = 0;
        image_back_mix[index * 3 + 2] = 0;
        return;
    }

    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;

    loc_prob += offset;
    style_rgbs += offset * 3;
    beck_rgbs += offset * 3;

    // accumulate
    uint32_t step = 0;

    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, t = 0, d = 0;
    scalar_t  localization_prob = 0;
    scalar_t r_stylization = 0, g_stylization = 0, b_stylization = 0;
    scalar_t r_background = 0, g_background = 0, b_background = 0;
    scalar_t r_yellow = 0.8, g_yellow = 1.0, b_yellow = 0.0;
    scalar_t r_gray = 0.71, g_gray = 0.71, b_gray = 0.71;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        localization_prob += weight * loc_prob[0];

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        r_stylization += weight * style_rgbs[0];
        g_stylization += weight * style_rgbs[1];
        b_stylization += weight * style_rgbs[2];

        r_background += weight * beck_rgbs[0];
        g_background += weight * beck_rgbs[1];
        b_background += weight * beck_rgbs[2];

        t += deltas[1]; // real delta
        d += weight * t;

        ws += weight;

        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;

        loc_prob++;
        style_rgbs += 3;
        beck_rgbs += 3;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;
    image_origin[index * 3] = r;
    image_origin[index * 3 + 1] = g;
    image_origin[index * 3 + 2] = b;

    image_prob[index] = localization_prob;

    image_style_origin[index * 3 ] = r_stylization;
    image_style_origin[index * 3  + 1] = g_stylization;
    image_style_origin[index * 3  + 2] = b_stylization;

    image_back_origin[index * 3 ] = r_background;
    image_back_origin[index * 3  + 1] = g_background;
    image_back_origin[index * 3  + 2] = b_background;

    image_localization_mix[index * 3 ] = localization_prob * r_yellow + (1 - localization_prob) * r_gray;
    image_localization_mix[index * 3  + 1] = localization_prob * g_yellow + (1 - localization_prob) * g_gray;
    image_localization_mix[index * 3  + 2] = localization_prob * b_yellow + (1 - localization_prob) * b_gray;
    
    image_style_mix[index * 3 ] = localization_prob * r_stylization + (1 - localization_prob) * r_gray;
    image_style_mix[index * 3  + 1] = localization_prob * g_stylization + (1 - localization_prob) * g_gray;
    image_style_mix[index * 3  + 2] = localization_prob * b_stylization + (1 - localization_prob) * b_gray;

    image_back_mix[index * 3 ] = localization_prob * r_yellow + (1 - localization_prob) * r_background;
    image_back_mix[index * 3  + 1] = localization_prob * g_yellow + (1 - localization_prob) * g_background;
    image_back_mix[index * 3  + 2] = localization_prob * b_yellow + (1 - localization_prob) * b_background;
}

void composite_rays_train_localization_int_forward(const at::Tensor sigmas, const at::Tensor rgbs, 
                                  const at::Tensor loc_prob, const at::Tensor style_rgbs, const at::Tensor beck_rgbs,
                                  const at::Tensor deltas, const at::Tensor rays, const uint32_t M, const uint32_t N,
                                  const float T_thresh, 
                                  at::Tensor weights_sum, at::Tensor depth, at::Tensor image_origin,
                                  at::Tensor image_prob, at::Tensor image_style_origin, at::Tensor image_back_origin,
                                  at::Tensor image_localization_mix, at::Tensor image_style_mix,
                                  at::Tensor image_back_mix) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_localization_int_forward", ([&] {
        kernel_composite_rays_train_localization_int_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(),
                                                                                     rgbs.data_ptr<scalar_t>(),
                                                                                     loc_prob.data_ptr<scalar_t>(),
                                                                                     style_rgbs.data_ptr<scalar_t>(),
                                                                                     beck_rgbs.data_ptr<scalar_t>(),
                                                                                     deltas.data_ptr<scalar_t>(),
                                                                                     rays.data_ptr<int>(),
                                                                                     M, N, T_thresh,
                                                                                     weights_sum.data_ptr<scalar_t>(),
                                                                                     depth.data_ptr<scalar_t>(),
                                                                                     image_origin.data_ptr<scalar_t>(),
                                                                                     image_prob.data_ptr<scalar_t>(),
                                                                                     image_style_origin.data_ptr<scalar_t>(),
                                                                                     image_back_origin.data_ptr<scalar_t>(),
                                                                                     image_localization_mix.data_ptr<scalar_t>(),
                                                                                     image_style_mix.data_ptr<scalar_t>(),
                                                                                     image_back_mix.data_ptr<scalar_t>());

        
    }));

}  


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_localization_int_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ grad_image_style,
    const scalar_t * __restrict__ grad_image_back,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,
    const scalar_t * __restrict__ loc_prob,
    const scalar_t * __restrict__ style_rgbs,
    const scalar_t * __restrict__ beck_rgbs,
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ image_origin,
    const scalar_t * __restrict__ image_prob,
    const scalar_t * __restrict__ image_style_origin,
    const scalar_t * __restrict__ image_back_origin,
    const scalar_t * __restrict__ image_localization_mix,
    const scalar_t * __restrict__ image_style_mix,
    const scalar_t * __restrict__ image_back_mix,
    const uint32_t M, const uint32_t N, const float T_thresh,
    scalar_t * grad_loc_prob,
    scalar_t * grad_rgbs_style,
    scalar_t * grad_rgbs_back
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights_sum += index;
    grad_image += index * 3;
    weights_sum += index;

    image_origin += index * 3;

    image_prob += index;

    image_style_origin += index * 3;

    image_back_origin += index * 3;

    image_localization_mix += index * 3;

    image_style_mix += index * 3;

    image_back_mix += index * 3;

    sigmas += offset;
    rgbs += offset * 3;
    loc_prob += offset;
    style_rgbs += offset * 3;
    beck_rgbs += offset * 3;
    deltas += offset * 2;
    grad_loc_prob += offset;
    grad_rgbs_style += offset * 3;
    grad_rgbs_back += offset * 3;

    // accumulate
    uint32_t step = 0;

    scalar_t T = 1.0f;
    // const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0];
    // scalar_t r = 0, g = 0, b = 0, ws = 0;
    scalar_t r_yellow = 0.8, g_yellow = 1.0, b_yellow = 0.0;
    scalar_t r_gray = 0.71, g_gray = 0.71, b_gray = 0.71;

    while (step < num_steps) {

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;

        T *= 1.0f - alpha;

        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs_style[0] = grad_image_style[0] * weight * image_prob[0];
        grad_rgbs_style[1] = grad_image_style[1] * weight * image_prob[0];
        grad_rgbs_style[2] = grad_image_style[2] * weight * image_prob[0];

        grad_rgbs_back[0] = grad_image_back[0] * weight * ( 1- image_prob[0]);
        grad_rgbs_back[1] = grad_image_back[1] * weight * ( 1- image_prob[0]);
        grad_rgbs_back[2] = grad_image_back[2] * weight * ( 1- image_prob[0]);

        // write grad_sigmas

        grad_loc_prob[0] = grad_image[0] * weight * (r_yellow - r_gray) +
                           grad_image[1] * weight * (g_yellow - g_gray) +
                           grad_image[2] * weight * (b_yellow - b_gray) +
                        //    grad_image[3] * (T * rgbs[3] - (newvar_final - newvar)) TODO - add newvar gradient
                           grad_image_style[0] * weight * (image_style_origin[0] - r_gray) +
                           grad_image_style[1] * weight * (image_style_origin[1] - g_gray) +
                           grad_image_style[2] * weight * (image_style_origin[2] - b_gray) +
                           // grad_image_style[3] * weight * (style_rgbs[3] - rgbs[3]) + TODO - add newvar gradient
                        //    grad_image_back[0] * weight * (rgbs[0] - beck_rgbs[0]) +
                        //    grad_image_back[1] * weight * (rgbs[1] - beck_rgbs[1]) +
                        //    grad_image_back[2] * weight * (rgbs[2] - beck_rgbs[2])+
                        //    grad_image_back[3] * weight * (rgbs[3] - beck_rgbs[3]);
                            grad_image_back[0] * weight * (r_yellow - image_back_origin[0]) +
                            grad_image_back[1] * weight * (g_yellow - image_back_origin[1]) +
                            grad_image_back[2] * weight * (b_yellow - image_back_origin[2]);
                           // grad_image_back[3] * weight * (rgbs[3] - beck_rgbs[3]); TODO - add newvar gradient

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;

        // locate
        sigmas++;
        deltas += 2;
        grad_loc_prob++;
        grad_rgbs_style += 3;
        grad_rgbs_back += 3;

        step++;
    }
}



void composite_rays_train_localization_int_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_image,
                                const at::Tensor grad_image_style, const at::Tensor grad_image_back,
                                const at::Tensor sigmas, const at::Tensor rgbs, 
                                const at::Tensor loc_prob, const at::Tensor style_rgbs, const at::Tensor beck_rgbs, 
                                const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, 
                                const at::Tensor image_origin, const at::Tensor image_prob,
                                const at::Tensor image_style_origin,
                                const at::Tensor image_back_origin, const at::Tensor image_localization_mix,
                                const at::Tensor image_style_mix,
                                const at::Tensor image_back_mix,
                                const uint32_t M, const uint32_t N, const float T_thresh,
                                at::Tensor grad_loc_prob, at::Tensor grad_rgbs_style, at::Tensor grad_rgbs_back
                                ) {





    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_localization_int_backward", ([&] {
        kernel_composite_rays_train_localization_int_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(),
                                                                                    grad_image.data_ptr<scalar_t>(),
                                                                                    grad_image_style.data_ptr<scalar_t>(),
                                                                                    grad_image_back.data_ptr<scalar_t>(),
                                                                                    sigmas.data_ptr<scalar_t>(), 
                                                                                    rgbs.data_ptr<scalar_t>(),
                                                                                    loc_prob.data_ptr<scalar_t>(),
                                                                                    style_rgbs.data_ptr<scalar_t>(),
                                                                                    beck_rgbs.data_ptr<scalar_t>(),
                                                                                    deltas.data_ptr<scalar_t>(), 
                                                                                    rays.data_ptr<int>(), 
                                                                                    weights_sum.data_ptr<scalar_t>(),
                                                                                    image_origin.data_ptr<scalar_t>(),
                                                                                    image_prob.data_ptr<scalar_t>(),
                                                                                    image_style_origin.data_ptr<scalar_t>(),
                                                                                    image_back_origin.data_ptr<scalar_t>(),
                                                                                    image_localization_mix.data_ptr<scalar_t>(),
                                                                                    image_style_mix.data_ptr<scalar_t>(),
                                                                                    image_back_mix.data_ptr<scalar_t>(),
                                                                                    M, N, T_thresh,    
                                                                                    grad_loc_prob.data_ptr<scalar_t>(),
                                                                                    grad_rgbs_style.data_ptr<scalar_t>(),
                                                                                    grad_rgbs_back.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_composite_rays_localization_int(
    const uint32_t n_alive,
    const uint32_t n_step,
    const float T_thresh,
    int* rays_alive,
    scalar_t* rays_t,
    const scalar_t* __restrict__ sigmas,
    const scalar_t* __restrict__ rgbs,
    const scalar_t* __restrict__ loc_prob,
    const scalar_t* __restrict__ style_rgbs,
    const scalar_t* __restrict__ beck_rgbs,
    const scalar_t* __restrict__ deltas,
    scalar_t* weights_sum, scalar_t* depth,
    scalar_t* image_origin,
    scalar_t* image_prob,
    scalar_t* image_style_origin,
    scalar_t* image_back_origin,
    scalar_t* image_localization_mix,
    scalar_t* image_style_mix,
    scalar_t* image_back_mix
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id

    // locate
    sigmas += n * n_step;
    loc_prob += n * n_step;
    rgbs += n * n_step * 3;
    style_rgbs += n * n_step * 3;
    beck_rgbs += n * n_step * 3;
    deltas += n * n_step * 2;

    rays_t += index;
    weights_sum += index;
    depth += index;
    image_origin += index * 3;
    image_prob += index;
    image_style_origin += index * 3;
    image_back_origin += index * 3;
    image_localization_mix += index * 3;
    image_style_mix += index * 3;
    image_back_mix += index * 3;

    scalar_t t = rays_t[0]; // current ray's t

    //TODO
    scalar_t r_yellow = 0.8, g_yellow = 1.0, b_yellow = 0.0;
    scalar_t r_gray = 0.71, g_gray = 0.71, b_gray = 0.71;

    //






    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t localization_prob = image_prob[0];
    scalar_t r = image_origin[0];
    scalar_t g = image_origin[1];
    scalar_t b = image_origin[2];
    
    scalar_t r_stylization = image_style_origin[0];
    scalar_t g_stylization = image_style_origin[1];
    scalar_t b_stylization = image_style_origin[2];

    scalar_t r_background = image_back_origin[0];
    scalar_t g_background = image_back_origin[1];
    scalar_t b_background = image_back_origin[2];

    // accumulate
    uint32_t step = 0;
    while (step < n_step) {

        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;

        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /*
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        -->
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t += deltas[1]; // real delta
        d += weight * t;


        r += weight * rgbs[0]; // TODO - ADD YELLOW COLOR red chanel
        g += weight * rgbs[1]; // TODO - ADD YELLOW COLOR green chanel
        b += weight * rgbs[2]; // TODO - ADD YELLOW COLOR blue chanel

        localization_prob += weight * loc_prob[0];

        r_stylization += weight * style_rgbs[0];
        g_stylization += weight * style_rgbs[1];
        b_stylization += weight * style_rgbs[2];


        r_background += weight * beck_rgbs[0];
        g_background += weight * beck_rgbs[1];
        b_background += weight * beck_rgbs[2];

        
        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        loc_prob++;
        style_rgbs += 3;
        beck_rgbs += 3;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;

    image_origin[0] = r;
    image_origin[1] = g;
    image_origin[2] = b;

    image_prob[0] = localization_prob;

    image_style_origin[0] = r_stylization;
    image_style_origin[1] = g_stylization;
    image_style_origin[2] = b_stylization;

    image_back_origin[0] = r_background;
    image_back_origin[1] = g_background;
    image_back_origin[2] = b_background;

    image_localization_mix[0] = localization_prob * r_yellow + (1 - localization_prob) * r_gray;
    image_localization_mix[1] = localization_prob * g_yellow + (1 - localization_prob) * g_gray;
    image_localization_mix[2] = localization_prob * b_yellow + (1 - localization_prob) * b_gray;
    
    image_style_mix[0] = localization_prob * r_stylization + (1 - localization_prob) * r_gray;
    image_style_mix[1] = localization_prob * g_stylization + (1 - localization_prob) * g_gray;
    image_style_mix[2] = localization_prob * b_stylization + (1 - localization_prob) * b_gray;

    image_back_mix[0] = localization_prob * r_yellow + (1 - localization_prob) * r_background;
    image_back_mix[1] = localization_prob * g_yellow + (1 - localization_prob) * g_background;
    image_back_mix[2] = localization_prob * b_yellow + (1 - localization_prob) * b_background;
}


void composite_rays_localization_int(const uint32_t n_alive, const uint32_t n_step, const float T_thresh,
                                 at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs,
                                 at::Tensor loc_prob, at::Tensor style_rgbs, at::Tensor beck_rgbs,
                                 at::Tensor deltas, at::Tensor weights, at::Tensor depth,
                                 at::Tensor image_origin, at::Tensor image_prob,
                                 at::Tensor image_style_origin,
                                 at::Tensor image_back_origin,
                                 at::Tensor image_localization_mix,
                                 at::Tensor image_style_mix,
                                 at::Tensor image_back_mix) {

    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image_origin.scalar_type(), "composite_rays_localization_int", ([&] {
        kernel_composite_rays_localization_int<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh,
                                                                                          rays_alive.data_ptr<int>(),
                                                                                          rays_t.data_ptr<scalar_t>(),
                                                                                          sigmas.data_ptr<scalar_t>(),
                                                                                          rgbs.data_ptr<scalar_t>(),
                                                                                          loc_prob.data_ptr<scalar_t>(),
                                                                                          style_rgbs.data_ptr<scalar_t>(),
                                                                                          beck_rgbs.data_ptr<scalar_t>(),
                                                                                          deltas.data_ptr<scalar_t>(),
                                                                                          weights.data_ptr<scalar_t>(),
                                                                                          depth.data_ptr<scalar_t>(),
                                                                                          image_origin.data_ptr<scalar_t>(),
                                                                                          image_prob.data_ptr<scalar_t>(),
                                                                                          image_style_origin.data_ptr<scalar_t>(),
                                                                                          image_back_origin.data_ptr<scalar_t>(),
                                                                                          image_localization_mix.data_ptr<scalar_t>(),
                                                                                          image_style_mix.data_ptr<scalar_t>(),
                                                                                          image_back_mix.data_ptr<scalar_t>());
    }));
}
